
#include <hip/hip_runtime.h>
#include <iostream>
#include <nccl.h>

const int kWarmUpTurns = 100;
const size_t kDataSize = 128 * 1024 * 1024; // 128 MB

void checkNcclError(ncclResult_t result, int line) {
    if (result != ncclSuccess) {
        std::cerr << "NCCL Error " << result << " at line " << line << ": " << ncclGetErrorString(result) << std::endl;
        exit(1);
    }
}
#define NCCL_CHECK(cmd) checkNcclError(cmd, __LINE__)

float P2PBandwidthTest(int device_id1, int device_id2, ncclComm_t *comms, ncclUniqueId id) {
    // Memory allocation
    float* sendBuffer;
    float* recvBuffer;

    hipStream_t s1 = (hipStream_t)malloc(sizeof(hipStream_t));
    hipStream_t s2 = (hipStream_t)malloc(sizeof(hipStream_t));

    // Allocate buffer and prepare stream for each gpu
    hipSetDevice(device_id1);
    hipMalloc(&sendBuffer, kDataSize * sizeof(float));
    hipMemset(sendBuffer, 1.0, kDataSize * sizeof(float));
    hipStreamCreate(&s1);

    hipSetDevice(device_id2);
    hipMalloc(&recvBuffer, kDataSize * sizeof(float));
    hipMemset(recvBuffer, 0.0, kDataSize * sizeof(float));
    hipStreamCreate(&s2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record elapsed time for rank n send and rank m recv
    hipEventRecord(start);
    ncclGroupStart();
    NCCL_CHECK(ncclSend(sendBuffer, kDataSize, ncclFloat, device_id2, comms[device_id1], s1));
    NCCL_CHECK(ncclRecv(recvBuffer, kDataSize, ncclFloat, device_id1, comms[device_id2], s2));
    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);
    ncclGroupEnd();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
 
    // Clean up
    hipSetDevice(device_id1);
    NCCL_CHECK(ncclCommDestroy(comms[device_id1]));
    hipSetDevice(device_id2);
    NCCL_CHECK(ncclCommDestroy(comms[device_id2]));

    hipSetDevice(device_id1);
    hipFree(sendBuffer);

    hipSetDevice(device_id2);
    hipFree(recvBuffer);

    return milliseconds / 1000.0; // Convert to seconds
    return 0.0;
}

int main() {
    int version;
    ncclGetVersion(&version);
    printf("nccl version: %d\n", version);

    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    printf("device count: %d\n", numGPUs);

    if (numGPUs < 2) {
        std::cout << "Error: At least two GPUs are required." << std::endl;
        return 0;
    }

    ncclComm_t comms[numGPUs];
    ncclUniqueId id;
    NCCL_CHECK(ncclGetUniqueId(&id));
    // create communicator for each GPU in a single node
    ncclCommInitAll(comms, numGPUs, NULL);
    // for(int i = 0; i < numGPUs; i++) {
    //     cudaSetDevice(i);
    //     ncclCommInitRank(&(comms[i]), numGPUs, id, i);
    // }

    for (int i = 0; i < numGPUs; i++) {
        for (int j = 0; j < numGPUs; j++) {
            float time = P2PBandwidthTest(i, j, comms, id);
            float bandwidth = (kDataSize * sizeof(float) / 1024.0 / 1024.0 / 1024.0) / time; // in GB/s
            printf("GPU %d -> GPU %d: Bandwidth: %10.2f GB/s\n", i, j, bandwidth);
        }
    }

    return 0;
}